#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../lib/coo.h"
#include "../lib/cm.h"
#include "../lib/mm.h"
/*{{{*/
#define ERROR_CHECK \
{\
	hipError_t err = hipGetLastError(); \
	if ( hipSuccess != err ) \
	{\
		printf("[%s:%d]CUDA ERROR : %s\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
		exit(-1); \
	}\
}
/*}}}*/

#define DENSE_NUM 1000

int* match;
int* num_splits;

int* a_ptr;
int* a_idx;
float* a_val;

int* b_ptr;
int* b_idx;
float* b_val;

int* c_ptr_nnz;
int* c_ptr_base;
int* c_idx;
int* c_idx_dummy;
float* c_val;

int* dense;


__device__ int a_num_cols, a_num_rows;
__device__ bool a_type;
__device__ int b_num_cols, b_num_rows;
__device__ bool b_type;
__device__ int c_num_cols, c_num_rows, c_nnz;
__device__ bool c_type;

__global__ void coo2cm(/*{{{*/
        triplet* data, int* nnz, bool* type,
        int* ptr, int *idx, float* val)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
    if(global_tid<*nnz)
    {
        if(*type == ROW_MAJOR)
        {
            idx[global_tid] = data[global_tid].cidx;
            val[global_tid] = data[global_tid].val;
            atomicAdd(&ptr[data[global_tid].ridx+1], 1);
        }
        else
        {
            idx[global_tid] = data[global_tid].ridx;
            val[global_tid] = data[global_tid].val;
            atomicAdd(&ptr[data[global_tid].cidx+1], 1);
        }
    }
}

cm cudaCOO2CM(coo mat)
{
    cm ret;
    cmInit(&ret);
    
    cmSetNNZ(&ret, mat.nnz);
    cmSetNumRows(&ret, mat.num_rows);
    cmSetNumCols(&ret, mat.num_cols);
    cmSetType(&ret, mat.type);

    triplet *data;
    bool *type;
    int *nnz;

    hipMalloc((void**)&data, sizeof(triplet)*mat.nnz);
    hipMalloc((void**)&type, sizeof(bool));
    hipMalloc((void**)&nnz, sizeof(int));
    hipMemcpy(data, mat.data, sizeof(triplet)*mat.nnz, hipMemcpyHostToDevice);
    hipMemcpy(type, &mat.type, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(nnz, &mat.nnz, sizeof(int), hipMemcpyHostToDevice);

    // CSC
    if(mat.type==COL_MAJOR)
    {
        hipMalloc((void**)&a_ptr, sizeof(int)*(ret.num_cols+1));
        hipMalloc((void**)&a_idx, sizeof(int)*(ret.nnz));
        hipMalloc((void**)&a_val, sizeof(float)*(ret.nnz));

        coo2cm<<< mat.nnz/128+1, 128>>>
            (data, nnz, type,
             a_ptr, a_idx, a_val);

        ret.ptr = new int[ret.num_cols+1];
        ret.idx = new int[ret.nnz];
        ret.val = new float[ret.nnz];
        hipMemcpy(ret.ptr, a_ptr, sizeof(int)*(ret.num_cols+1),hipMemcpyDeviceToHost);
        hipMemcpy(ret.idx, a_idx, sizeof(int)*ret.nnz,hipMemcpyDeviceToHost);
        hipMemcpy(ret.val, a_val, sizeof(float)*ret.nnz,hipMemcpyDeviceToHost);

        ret.ptr[0] = 0;
        for(int i=0; i<ret.num_cols;i++){
            ret.ptr[i+1] += ret.ptr[i];
        }
        hipMemcpy(a_ptr, ret.ptr, sizeof(int)*(ret.num_cols+1),hipMemcpyHostToDevice);
    }
    else{
        hipMalloc((void**)&b_ptr, sizeof(int)*(ret.num_rows+1));
        hipMalloc((void**)&b_idx, sizeof(int)*(ret.nnz));
        hipMalloc((void**)&b_val, sizeof(float)*(ret.nnz));

        coo2cm<<< mat.nnz/128+1, 128>>>
            (data,nnz,type,
             b_ptr, b_idx, b_val);

        ret.ptr = new int[ret.num_rows+1];
        ret.idx = new int[ret.nnz];
        ret.val = new float[ret.nnz];
        hipMemcpy(ret.ptr, b_ptr, sizeof(int)*(ret.num_rows+1),hipMemcpyDeviceToHost);
        hipMemcpy(ret.idx, b_idx, sizeof(int)*ret.nnz,hipMemcpyDeviceToHost);
        hipMemcpy(ret.val, b_val, sizeof(float)*ret.nnz,hipMemcpyDeviceToHost);

        ret.ptr[0] = 0;
        for(int i=0; i<ret.num_rows;i++){
            ret.ptr[i+1] += ret.ptr[i];
        }
        hipMemcpy(b_ptr, ret.ptr, sizeof(int)*(ret.num_cols+1),hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();
    hipFree(data);
    hipFree(type);
    return ret;
}/*}}}*/

__global__ void initGEMM(
        int* a_ptr, int* a_idx,
        int* b_ptr, int* b_idx,
        int *c_ptr_base)
{
    for(int ai = a_ptr[blockIdx.x] + threadIdx.x; ai < a_ptr[blockIdx.x+1]; ai+= blockDim.x)
    {
        int row = a_idx[ai];
        atomicAdd(&c_ptr_base[row+1], b_ptr[blockIdx.x+1] - b_ptr[blockIdx.x]);
    }
}

cm cudaInitGEMM(cm A, cm B)/*{{{*/
{
    cm C;
    cmSetType(&C, ROW_MAJOR);
    cmSetNumRows(&C, cmGetNumRows(A));
    cmSetNumCols(&C, cmGetNumCols(B));

    hipMemcpyToSymbol(HIP_SYMBOL(a_num_cols), &A.num_cols, sizeof(int),0 ,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(a_num_rows), &A.num_rows, sizeof(int),0 ,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(b_num_cols), &B.num_cols, sizeof(int),0 ,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(b_num_rows), &B.num_rows, sizeof(int),0 ,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_num_cols), &C.num_cols, sizeof(int),0 ,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_num_rows), &C.num_rows, sizeof(int),0 ,hipMemcpyHostToDevice);

    hipMalloc((void**)&c_ptr_base,sizeof(int)*(cmGetNumRows(C)+1));
    hipMalloc((void**)&c_ptr_nnz,sizeof(int)*cmGetNumRows(C));
    ERROR_CHECK;
    initGEMM<<< A.num_cols, 32 >>>
        (a_ptr, a_idx,
         b_ptr, b_idx,
         c_ptr_base);
	hipDeviceSynchronize();

    int *temp = new int[cmGetNumRows(C)+1];
    hipMemcpy(temp, c_ptr_base, sizeof(int)*(cmGetNumRows(C)+1), hipMemcpyDeviceToHost);

    for(int i=0;i<A.num_rows;i++)
    {
        temp[i+1] += temp[i];
    }
    cmSetNNZ(&C,temp[A.num_rows]);
    printf("%d %d\n",cmGetNNZ(C),temp[A.num_rows]);
    
    hipMalloc((void**)&c_idx, sizeof(int)*cmGetNNZ(C));
    hipMalloc((void**)&c_val, sizeof(float)*cmGetNNZ(C));
    hipMemcpy(c_ptr_base, temp,sizeof(int)*(cmGetNumRows(C)+1), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_nnz), &C.nnz, sizeof(int),0 ,hipMemcpyHostToDevice);

    hipMemset(c_ptr_nnz, 0, sizeof(int)*cmGetNumRows(C));
	hipDeviceSynchronize();

    delete temp;
    return C;
}
/*}}}*/

__global__ void inspectGEMM(/*{{{*/
        int* a_ptr, int* b_ptr, int* counter)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < a_num_cols){
        int a_len = a_ptr[idx+1] - a_ptr[idx];
        int b_len = b_ptr[idx+1] - b_ptr[idx];
        int workload = a_len * b_len;

        if(workload > c_nnz/a_num_cols*50)
        {
            atomicAdd(&counter[0],1);
        }
        else
        {
            int left = 1;
            int right = 2;
            bool flag = 0;
            for(int i=1;i<8;i++)
            {
                if(left<= b_len && b_len<right)
                {
                    flag = 1;
                    atomicAdd(&counter[i],1);
                }
                left<<=1; 
                right<<=1;
            }
            if(!flag)
            {
                atomicAdd(&counter[8],1);
            }
        }
    }
}

__global__ void categorizeGEMM(
        int* a_ptr, int* b_ptr, 
        int* counter, int* bin)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx<a_num_cols){
        int a_len = a_ptr[idx+1] - a_ptr[idx];
        int b_len = b_ptr[idx+1] - b_ptr[idx];
        int workload = a_len * b_len;

        if(workload > c_nnz/a_num_cols*50)
        {
            int loc = atomicAdd(&counter[0],1);
            bin[loc] = idx;
        }
        else
        {
            int left = 1;
            int right = 2;
            bool flag = 0;
            for(int i=1;i<8;i++)
            {
                if(left<= b_len && b_len<right)
                {
                    int loc = atomicAdd(&counter[i],1);
                    flag = 1;
                    bin[loc] = idx;
                }
                left<<=1; 
                right<<=1;
            }
            if(!flag)
            {
                int loc = atomicAdd(&counter[8],1);
                bin[loc] = idx;
            }

        }
    }
}

int* t_counter;
int* counter;
int* bin;
#define NUM_BINS 9
void cudaCategorizeGEMM(cm A, cm B)
{
    hipMalloc((void**)&counter, sizeof(int)*(NUM_BINS+1));
    hipMalloc((void**)&bin, sizeof(int)*cmGetNumCols(A));
    hipMemset(counter, 0, sizeof(int)*(NUM_BINS+1));

    inspectGEMM<<< cmGetNumCols(A)/32+1, 32 >>>
        (a_ptr, b_ptr, counter);
    
    t_counter=new int[10];

    hipMemcpy(&t_counter[1], counter, sizeof(int)*NUM_BINS,hipMemcpyDeviceToHost);

    t_counter[0] = 0;
    for(int i=0; i < 9;i++)
    {
        t_counter[i+1] += t_counter[i];
    }
    
    hipMemcpy(counter, t_counter, sizeof(int)*(NUM_BINS+1),hipMemcpyHostToDevice);
    categorizeGEMM<<< cmGetNumCols(A)/32+1, 32>>>
        (a_ptr, b_ptr, counter, bin);
    ERROR_CHECK;
}/*}}}*/

#define WIDTH 4/*{{{*/
__global__ void splitGEMM(
    int* a_ptr, int* a_idx, float* a_val,
    int* b_ptr, int* b_idx, float* b_val,
    int* c_ptr_base, int* c_ptr_nnz, int* c_idx, float* c_val,
    int* match, int* num_splits,
    int*bin)
{
    __shared__ int offset;
    int idx = match[blockIdx.x];
    int bid = blockIdx.x - num_splits[blockIdx.x];

    int b_curr = b_ptr[idx];
    int b_next = b_ptr[idx+1];
    for(int ai = a_ptr[idx]+bid*WIDTH; ai < a_ptr[idx+1] && ai< a_ptr[idx] + (bid+1)*WIDTH; ai++)
    {
        int row = a_idx[ai];
        int base = c_ptr_base[row];
        if(threadIdx.x==0)
            offset = atomicAdd(&c_ptr_nnz[row], b_next-b_curr);
        __syncthreads();
        for(int bi = threadIdx.x; bi < b_next - b_curr; bi+=blockDim.x)
        {
            c_val[base+offset+bi] = a_val[ai]*b_val[b_curr+bi];
            c_idx[base+offset+bi] = b_idx[b_curr+bi];
        }
    }
}

void cudaSplitGEMM(cm A, cm B, cm C)
{   

    int orig = t_counter[1] - t_counter[0];
    int* t_bin = new int[orig];
    int* t_match = new int[cmGetNumRows(A)];
    int* t_num_splits = new int[cmGetNumRows(A)];

    hipMemcpy(t_bin, bin, sizeof(int)*orig,hipMemcpyDeviceToHost);
    int num_blocks = 0;
    int prev=0;
    for(int i=0;i<orig;i++)
    {
        int idx = t_bin[i];
        int len =  A.ptr[idx+1] - A.ptr[idx];
        int q = len / WIDTH;
        int r = len % WIDTH;
        int split = r?q+1:q;

        for(int j=0;j<split;j++)
        {
            t_num_splits[num_blocks] = prev;
            t_match[num_blocks++] = idx;
        }
        prev += split;
    }
    hipMalloc((void**)&match, sizeof(int)*num_blocks);
    hipMalloc((void**)&num_splits, sizeof(int)*num_blocks);
    hipMemcpy(match, t_match, sizeof(int)*num_blocks,hipMemcpyHostToDevice);
    hipMemcpy(num_splits, t_num_splits, sizeof(int)*num_blocks,hipMemcpyHostToDevice);
    splitGEMM<<<num_blocks, 32 >>>
            (a_ptr, a_idx, a_val,
             b_ptr, b_idx, b_val,
             c_ptr_base, c_ptr_nnz , c_idx, c_val,
             match, num_splits,
             &bin[t_counter[0]]);

}
/*}}}*/

__global__ void binGEMM(
        int* a_ptr, int* a_idx, float*a_val,
        int* b_ptr, int* b_idx, float*b_val,
        int* c_ptr_base, int* c_ptr_nnz, int* c_idx, float* c_val,
        int* bin)
{
    __shared__ int offset;
    int idx = bin[blockIdx.x];

    int b_curr = b_ptr[idx];
    int b_next = b_ptr[idx+1];
    for(int ai = a_ptr[idx]; ai < a_ptr[idx+1]; ai++)
    {
        int row = a_idx[ai];
        int base = c_ptr_base[row];
        if(threadIdx.x==0)
            offset = atomicAdd(&c_ptr_nnz[row], b_next-b_curr);
        __syncthreads();
        for(int bi = threadIdx.x; bi < b_next - b_curr; bi+=blockDim.x)
        {
            c_val[base+offset+bi] = a_val[ai]*b_val[b_curr+bi];
            c_idx[base+offset+bi] = b_idx[b_curr+bi];
        }
    }
}

void cudaBinGEMM(cm A, cm B, cm C)
{    
    for(int i=1;i<NUM_BINS;i++){
        int num_blocks = t_counter[i+1] - t_counter[i];
        if(num_blocks)
        binGEMM<<<num_blocks, 128 >>>
            (a_ptr, a_idx, a_val,
             b_ptr, b_idx, b_val,
             c_ptr_base, c_ptr_nnz , c_idx, c_val,
             &bin[t_counter[i]]);
    }
    hipDeviceSynchronize();
    ERROR_CHECK;
}




__global__ void simpleGEMM(/*{{{*/
        int* a_ptr, int* a_idx, float*a_val,
        int* b_ptr, int* b_idx, float*b_val,
        int*c_ptr_base, int* c_ptr_nnz, int* c_idx, float* c_val)
{
    __shared__ int offset;

    int b_curr = b_ptr[blockIdx.x];
    int b_next = b_ptr[blockIdx.x+1];
    for(int ai = a_ptr[blockIdx.x]; ai < a_ptr[blockIdx.x+1]; ai++)
    {
        int row = a_idx[ai];
        int base = c_ptr_base[row];
        if(threadIdx.x==0)
            offset = atomicAdd(&c_ptr_nnz[row], b_next-b_curr);
        __syncthreads();
        for(int bi = threadIdx.x; bi < b_next - b_curr; bi+=blockDim.x)
        {
            c_val[base+offset+bi] = a_val[ai]*b_val[b_curr+bi];
            c_idx[base+offset+bi] = b_idx[b_curr+bi];
        }
    }
}

void cudaSimpleGEMM(cm A, cm B, cm C)
{    

    simpleGEMM<<<cmGetNumCols(A), 32 >>>
        (a_ptr, a_idx, a_val,
         b_ptr, b_idx, b_val,
         c_ptr_base, c_ptr_nnz , c_idx, c_val);
    hipDeviceSynchronize();
}
/*}}}*/
__global__ void mergeGEMM(/*{{{*/
        int* c_ptr_base, int* c_ptr_nnz, int* c_idx, float* c_val,
        int* c_idx_dummy,int* dense
        )
{

    __shared__ int sh_loc;

    for(int r = 0; r <= c_num_rows/gridDim.x + 1 ; r++)
    {
        __syncthreads();
        sh_loc = 0;
        __syncthreads();
        int bid = (blockIdx.x + r*gridDim.x);
        int d_base = (blockIdx.x * c_num_cols);
        int c_base = c_ptr_base[bid];

        for(int ci = threadIdx.x ; ci < c_ptr_nnz[bid]; ci+= blockDim.x)
        {
            int col = c_idx_dummy[c_base+ci];
            float is_zero = atomicAdd(&dense[d_base + col], c_val[c_base+ci]);
            if(is_zero<0.0001 && is_zero>-0.0001)
            {
                int loc = atomicAdd(&sh_loc, 1);
                c_idx[c_base + loc] = col;
            }
        }
        __syncthreads();
        for(int ci = threadIdx.x ; ci<sh_loc;ci+= blockDim.x)
        {
            int col = c_idx[c_base + ci];
            c_val[c_base + ci] = dense[d_base + col];
            atomicExch(&dense[d_base+col], 0);
        } 
        __syncthreads();
        if(threadIdx.x==0)
            c_ptr_nnz[bid] = sh_loc;
    }
}


void cudaMergeGEMM(cm C)
{
    int num_dense = DENSE_NUM;
    hipMalloc((void**)&c_idx_dummy, sizeof(int)*cmGetNNZ(C));
    hipMalloc((void**)&dense, sizeof(float)*cmGetNumCols(C)*num_dense);
    hipMemset(dense, 0,sizeof(float)*cmGetNumCols(C)*num_dense);
    hipMemcpy(c_idx_dummy, c_idx, sizeof(int)*cmGetNNZ(C),hipMemcpyDeviceToDevice);

    mergeGEMM<<<DENSE_NUM,64>>>(
            c_ptr_base, c_ptr_nnz, c_idx, c_val, c_idx_dummy,
            dense);
    hipDeviceSynchronize();

    int* t_ptr_nnz = new int[cmGetNumRows(C)];
    int* t_ptr_base = new int[cmGetNumRows(C)+1];
    int* t_idx = new int[cmGetNNZ(C)];
    float* t_val = new float[cmGetNNZ(C)];
    hipMemcpy(t_ptr_base, c_ptr_base, sizeof(int)*cmGetNumRows(C), hipMemcpyDeviceToHost);
    hipMemcpy(t_ptr_nnz, c_ptr_nnz, sizeof(int)*cmGetNumRows(C), hipMemcpyDeviceToHost);

    int nnzC=0;
    for(int i=0; i<cmGetNumRows(C);i++)
    {
        int c_base = t_ptr_base[i];
        int c_nnz = t_ptr_nnz[i];
        nnzC += c_nnz;
        hipMemcpy(&t_idx[c_base], &c_idx[c_base], sizeof(int)*c_nnz, hipMemcpyDeviceToHost);
        hipMemcpy(&t_val[c_base], &c_val[c_base], sizeof(float)*c_nnz, hipMemcpyDeviceToHost);
    }
    ERROR_CHECK;

    int nnzCf = 0;
    for(int i=0;i< cmGetNumRows(C);i++)
    {
        int c_base = t_ptr_base[i];
        int c_nnz = t_ptr_nnz[i];
        for(int j=0;j<c_nnz;j++)
        {
            if(t_val[c_base+j]>0.0001)
                nnzCf++;
        }
    }
    printf("%d %d\n",nnzC, nnzCf);
}/*}}}*/
